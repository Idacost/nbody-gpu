#include <hip/hip_runtime.h>
#include <cmath>

__global__ void calculate_forces(double* mass, double* x, double* y, double* z,
                                  double* fx, double* fy, double* fz, size_t nbpart, double G) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nbpart) {
        double Fx = 0.0, Fy = 0.0, Fz = 0.0;

        for (size_t j = 0; j < nbpart; ++j) {
            if (i != j) {
                double dx = x[j] - x[i];
                double dy = y[j] - y[i];
                double dz = z[j] - z[i];
                double dist_sq = dx * dx + dy * dy + dz * dz;
                double dist = sqrt(dist_sq);

                // Gravitational force formula
                double F = G * mass[i] * mass[j] / dist_sq;
                Fx += F * (dx / dist);
                Fy += F * (dy / dist);
                Fz += F * (dz / dist);
            }
        }

        // Store the net forces on particle i
        fx[i] = Fx;
        fy[i] = Fy;
        fz[i] = Fz;
    }
}

__global__ void update_positions_velocities(double* mass, double* x, double* y, double* z,
                                            double* vx, double* vy, double* vz,
                                            double* fx, double* fy, double* fz, size_t nbpart, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nbpart) {
        // Update velocity
        vx[i] += (fx[i] / mass[i]) * dt;
        vy[i] += (fy[i] / mass[i]) * dt;
        vz[i] += (fz[i] / mass[i]) * dt;

        // Update position
        x[i] += vx[i] * dt;
        y[i] += vy[i] * dt;
        z[i] += vz[i] * dt;
    }
}
